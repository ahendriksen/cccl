#include "hip/hip_runtime.h"

#include<cuda/atomic>

template <typename T, typename V> union U { T t; V v; };
using atom_t = cuda::atomic<int, cuda::thread_scope_device>*;
using aref_t = cuda::atomic_ref<int, cuda::thread_scope_device>;

// Type your code here, or load an example.
__global__ void square(int* data,
                       int* array,
                       int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n) {
        asm volatile("// Before atom_{ref} definition" ::: "memory");
        // Spill to  local happens here (for atomic_ref). (st.local)
        #ifdef AREF
        auto ref = aref_t{*(data + tid)};
        #else
        auto& ref = *U<atom_t, aref_t>{ .v = aref_t{*(data + tid)} }.t;
        #endif
        asm volatile("// After atom_{ref} definition" ::: "memory");

        ref.compare_exchange_strong(array[tid], tid, cuda::std::memory_order_acquire);
    }
}
